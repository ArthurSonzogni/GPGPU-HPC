#include "hip/hip_runtime.h"
#include "Simulator.hpp"
#include <cstdlib>
#include <iostream>
//#include "ProgressBar.hpp"
#include <fstream>
#include <sstream>
#include "Kernels.hpp"

#include "../../Common/src/ProgressBar.hpp"

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

float randDouble()
{ 
	return rand() / float(RAND_MAX);
}

Simulator::Simulator(
		int agent,
		int step,
		float wc, float wa, float ws,
		float rc, float ra, float rs,
        bool write):
	agent(agent),step(step),
	wc(wc),wa(wa),ws(ws),
	rc(rc),ra(ra),rs(rs),
    write(write)
{
	init();
}


void Simulator::init()
{
	float maxRadius = std::max(std::max(ra,rs),rc);
	gridSize = 1;
	cellSize = 1.0;
	// TODO Modify the kernel to allow more than 512 cells
	while(cellSize > maxRadius && gridSize <= 8)
	{
		gridSize += 1;
		cellSize = (float)1/gridSize;
	}
	gridSize -= 1;
	cellSize = (float)1/gridSize;

	// initialization initial position
	position.reserve(3*agent);
	for(int i = 0; i < agent; ++i)
	{
		position.push_back(randDouble()); // x
		position.push_back(randDouble()); // y
		position.push_back(randDouble()); // z
	}

	gpuCheck(hipGetLastError());

	// create the cuda data
	gpuCheck(hipMalloc((void**)&position_cuda,position.size()*sizeof(float)));
	gpuCheck(hipMalloc((void**)&speed_cuda,position.size()*sizeof(float)));
	gpuCheck(hipMalloc((void**)&speedIncrement_cuda,position.size()*sizeof(float)));

	gpuCheck(hipMalloc((void**)&cellFirst_cuda,gridSize*gridSize*gridSize*sizeof(int)));
	gpuCheck(hipMalloc((void**)&cellLast_cuda,gridSize*gridSize*gridSize*sizeof(int)));
	gpuCheck(hipMalloc((void**)&cellCount_cuda,gridSize*gridSize*gridSize*sizeof(int)));
	gpuCheck(hipMalloc((void**)&cellNeighbors_cuda,27*gridSize*gridSize*gridSize*sizeof(int)));
	gpuCheck(hipMalloc((void**)&cellDimension_cuda,6*gridSize*gridSize*gridSize*sizeof(float)));

	gpuCheck(hipMalloc((void**)&boidNext_cuda,agent*sizeof(int)));
	gpuCheck(hipMalloc((void**)&boidPrevious_cuda,agent*sizeof(int)));
	gpuCheck(hipMalloc((void**)&boidCell_cuda,agent*sizeof(int)));

	// copy the position to cuda
	gpuCheck(hipMemcpy(position_cuda, &(position[0]), 3*agent*sizeof(float), hipMemcpyHostToDevice));

	// init speed to zero
    int blockDim;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockDim, initToZero, 0, 3*agent);
    int gridDim = (3*agent + blockDim - 1) / blockDim; 
    initToZero<<<gridDim,blockDim>>>(speed_cuda, 3*agent);
    gpuCheck(hipGetLastError());

	// init lists & neighbors
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockDim, initCells, 0, gridSize*gridSize*gridSize);
    gridDim = (gridSize*gridSize*gridSize + blockDim - 1) / blockDim; 
	initCells<<<gridDim,blockDim>>>(cellFirst_cuda, cellLast_cuda, cellNeighbors_cuda, cellCount_cuda, cellDimension_cuda, cellSize, gridSize, position_cuda, boidNext_cuda, boidPrevious_cuda, boidCell_cuda, agent);
}


void Simulator::run()
{
    ProgressBar progressBar;
	for(int i = 0; i < step; ++i)
	{
		oneStep();
		progressBar.update(i/float(step));
		gpuCheck(hipMemcpy(&(position[0]), position_cuda, 3*agent*sizeof(float), hipMemcpyDeviceToHost));

		// print the result
		std::stringstream filename;
		filename << "./output/boids_" << i << ".xyz";
		if (write)  save(filename.str()); 
	}
}

void Simulator::oneStep()
{

    int blockDim,minGridSize,gridDim,dataSize;

    // computeSpeedIncrement
    dataSize = agent;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockDim, initToZero, 0, dataSize);
    gridDim = (dataSize + blockDim - 1) / blockDim; 
	computeSpeedIncrement<<<blockDim,gridDim>>>(position_cuda, speed_cuda, speedIncrement_cuda, boidNext_cuda, boidCell_cuda, dataSize, rs,ra,rc, ws,wa,wc, cellFirst_cuda, cellNeighbors_cuda);
	hipDeviceSynchronize();

    gpuCheck(hipGetLastError());

    // updatePosition
    dataSize = agent;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockDim, initToZero, 0, dataSize);
    gridDim = (dataSize + blockDim - 1) / blockDim; 
	updateSpeedPosition<<<blockDim,gridDim>>>(position_cuda, speed_cuda, speedIncrement_cuda, dataSize);
    gpuCheck(hipGetLastError());

	// updateList
	gridDim = 1;
	blockDim = gridSize*gridSize*gridSize;
	updateLists<<<gridDim,blockDim>>>(cellFirst_cuda, cellLast_cuda, cellNeighbors_cuda, cellCount_cuda, cellDimension_cuda, cellSize, gridSize, position_cuda, boidNext_cuda, boidPrevious_cuda, boidCell_cuda, agent);
}

void Simulator::save(const std::string& filename)
{
	std::ofstream file;
	file.open(filename.c_str());

	for(int i = 0; i < agent; ++i)
	{
		file
			<< position[3*i] << " "
			<< position[3*i+1] << " "
			<< position[3*i+2]
			<< std::endl;
	}

	file.close();
}
