#include "hip/hip_runtime.h"
#include "Simulator.hpp"
#include <cstdlib>
#include <iostream>
//#include "ProgressBar.hpp"
#include <fstream>
#include <sstream>
#include "Kernels.hpp"

#include "../../Common/src/ProgressBar.hpp"

#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

float randDouble()
{ 
	return rand() / float(RAND_MAX);
}

Simulator::Simulator(
		int agent,
		int step,
		float wc, float wa, float ws,
		float rc, float ra, float rs,
        bool write):
	agent(agent),step(step),
	wc(wc),wa(wa),ws(ws),
	rc(rc),ra(ra),rs(rs),
    write(write)
{
	init();
}


void Simulator::init()
{
	// initialization initial position
	position.reserve(3*agent);
	for(int i = 0; i < agent; ++i)
	{
		position.push_back(randDouble()); // x
		position.push_back(randDouble()); // y
		position.push_back(randDouble()); // z
	}

	gpuCheck(hipGetLastError());

	// create the cuda data
	gpuCheck(hipMalloc((void**)&position_cuda,position.size()*sizeof(float)));
	gpuCheck(hipMalloc((void**)&speed_cuda,position.size()*sizeof(float)));
	gpuCheck(hipMalloc((void**)&speedIncrement_cuda,position.size()*sizeof(float)));

	// copy the position to cuda
	gpuCheck(hipMemcpy(position_cuda, &(position[0]), 3*agent*sizeof(float), hipMemcpyHostToDevice));

	// init speed to zero
    int blockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initToZero, 0, 3*agent);
    int gridSize = (3*agent + blockSize - 1) / blockSize; 
    initToZero<<<gridSize,blockSize>>>(speed_cuda, 3*agent);
    gpuCheck(hipGetLastError());
}


void Simulator::run()
{
    ProgressBar progressBar;
	for(int i = 0; i < step; ++i)
	{
		oneStep();
		progressBar.update(i/float(step));
		gpuCheck(hipMemcpy(&(position[0]), position_cuda, 3*agent*sizeof(float), hipMemcpyDeviceToHost));

		// print the result
		std::stringstream filename;
		filename << "./output/boids_" << i << ".xyz";
		if (write)  save(filename.str()); 
	}
}

void Simulator::oneStep()
{

    int blockSize,minGridSize,gridSize,dataSize;

    // computeSpeedIncrement
    dataSize = agent;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initToZero, 0, dataSize);
    gridSize = (dataSize + blockSize - 1) / blockSize; 
	computeSpeedIncrement<<<blockSize,gridSize>>>(position_cuda, speed_cuda, speedIncrement_cuda, dataSize, rs,ra,rc, ws,wa,wc);
    gpuCheck(hipGetLastError());

    // computeSpeedIncrement
    dataSize = 3 * agent;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initToZero, 0, dataSize);
    gridSize = (dataSize + blockSize - 1) / blockSize; 
	updateSpeedPosition<<<blockSize,gridSize>>>(position_cuda, speed_cuda, speedIncrement_cuda, dataSize);
    gpuCheck(hipGetLastError());

    /*dim3 gridSize(1,1,1);*/
    /*dim3 blockSize(8,8,1);*/
	//	// compute the speedIncrement
	//	for(int i = 0; i < agent; ++i)
	//	{
	//		glm::dvec3 speedA(0.0),speedS(0.0),speedC(0.0);
	//		float countA=0,countS=0,countC=0;
	//		for(int j = 0; j < agent; ++j)
	//		{
	//			if(i == j) continue;
	//			glm::dvec3 direction = position[j] - position[i];
	//			float dist = glm::length(direction);
	//
	//			// separation/alignment/cohesion
	//			if (dist < rs )
	//			{
	//				speedS -= direction * ws;
	//				countS++;
	//			}
	//			if (dist < ra )
	//			{
	//				speedA += speed[j]  * wa;
	//				countA++;
	//			}
	//			if (dist < rc )
	//			{
	//				speedC += direction * wc;
	//				countC++;
	//			}
	//		}
	//		speedC = countC>0?speedC/countC:speedC;
	//		speedA = countA>0?speedA/countA:speedA;
	//		speedS = countS>0?speedS/countS:speedS;
	//		speedIncrement[i] = speedC+speedA+speedS;
	//	}
	//
	//	// sum the speedIncrement to the speed
	//	for(int i = 0; i < agent; ++i)
	//	{
	//		speed[i] += speedIncrement[i];
	//
	//		// limit the speed;
	//		const float maxSpeed = 0.3;
	//		float s = glm::length(speed[i]);
	//		if (s>maxSpeed)
	//			speed[i] *= maxSpeed/s;
	//	}
	//
	//	// sum the speed to the position (Euler intégration)
	//	for(int i = 0; i < agent; ++i)
	//	{
	//		position[i] += speed[i];
	////		position[i] = glm::modf(position[i], bounds);
	//		position[i] = glm::fract(position[i]);
	//	}
}

void Simulator::save(const std::string& filename)
{
	std::ofstream file;
	file.open(filename.c_str());

	for(int i = 0; i < agent; ++i)
	{
		file
			<< position[3*i] << " "
			<< position[3*i+1] << " "
			<< position[3*i+2]
			<< std::endl;
	}

	file.close();
}
